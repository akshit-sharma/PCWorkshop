
#include "hip/hip_runtime.h"


#include <stdio.h>

#include <time.h>
#include <stdlib.h>

#define ARR_SIZE 102400
#define THREADS 512

#define ARR_BYTE sizeof(int) * ARR_SIZE


__global__ void gpuSort(int * d_arr, size_t maxSize);


int main(int argv, char ** argc)
{
	int * h_arr;
	int * d_arr;
	int temp;
	int blockSize;
	size_t i;
	hipEvent_t start, stop;
	float milliseconds;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	h_arr = (int *)malloc(ARR_BYTE);

	hipMalloc((void **)&d_arr, ARR_BYTE);

	for (i = 0; i < ARR_SIZE; i++)
		h_arr[i] = rand() % 1024;

	hipMemcpy(d_arr, h_arr, ARR_BYTE, hipMemcpyHostToDevice);

	blockSize = ARR_SIZE / THREADS;
	blockSize += (ARR_SIZE%THREADS ? 1 : 0);
	blockSize += (blockSize / 2) + (blockSize % 2);

	hipEventRecord(start);
	for (i = 0; i < ARR_SIZE/2+1; i++) {
		gpuSort <<< blockSize, THREADS >>> (d_arr, ARR_SIZE);
	}
	hipEventRecord(stop);

	hipMemcpy(h_arr, d_arr, ARR_BYTE, hipMemcpyDeviceToHost);

	hipEventSynchronize(stop);

	for (i = 1; i < ARR_SIZE; i++) {
		if (h_arr[i - 1] > h_arr[i]) {
			printf("\nNot sorted\n\n");
			break;
		}
	}

	if (i == ARR_SIZE) {
		printf("\nSorted\n");
		hipEventElapsedTime(&milliseconds, start, stop);
		printf("Time taken : %llf\n", milliseconds / 1000);
	}

	hipFree(d_arr);

	free(h_arr);

}

__global__ void gpuSort(int * d_arr, size_t maxSize)
{
	int temp;
	size_t threadIndex = threadIdx.x + blockDim.x*blockIdx.x;

	threadIndex *= 2;

	if (threadIndex + 1 < maxSize) {
		if (d_arr[threadIndex] > d_arr[threadIndex + 1]) {
			temp = d_arr[threadIndex];
			d_arr[threadIndex] = d_arr[threadIndex + 1];
			d_arr[threadIndex + 1] = temp;
		}
	}
	threadIndex++;
	if (threadIndex + 1 < maxSize) {
		if (d_arr[threadIndex] > d_arr[threadIndex + 1]) {
			temp = d_arr[threadIndex];
			d_arr[threadIndex] = d_arr[threadIndex + 1];
			d_arr[threadIndex + 1] = temp;
		}
	}
	threadIndex--;
	if (threadIndex + 1 < maxSize) {
		if (d_arr[threadIndex] > d_arr[threadIndex + 1]) {
			temp = d_arr[threadIndex];
			d_arr[threadIndex] = d_arr[threadIndex + 1];
			d_arr[threadIndex + 1] = temp;
		}
	}


}

