
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define THREADS 128

__global__ void square(float * d_output, float * d_input)
{

	float value;
	size_t index;
	
	index = threadIdx.x;
	value = d_input[index];

	d_output[index] = value * value;

}


int main(int argc, char ** argv)
{

	float * h_array;
	float * d_input;
	float * d_output;

	size_t totalArraySize;

	totalArraySize = sizeof(float) * THREADS;

	h_array = (float *) malloc(totalArraySize);

	for (size_t i = 0; i < THREADS; i++)
		h_array[i] = i;

	hipMalloc((void **)&d_input, totalArraySize);
	hipMalloc((void **)&d_output, totalArraySize);

	hipMemcpy(d_input, h_array, totalArraySize, hipMemcpyHostToDevice);

	square <<<1, THREADS>>> (d_output, d_input);

	hipMemcpy(h_array, d_output, totalArraySize, hipMemcpyDeviceToHost);

	hipFree(d_input);
	hipFree(d_output);

	for (size_t i = 0; i < THREADS; i++) {
		printf("%8.3f", h_array[i]);
		printf(((i%7)!=6)?"\t":"\n");
	}
	printf("\n");

	free(h_array);

	return 0;

}


