#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define THREADS 128

__global__ void square(float * d_arr)
{

	float value;
	size_t index;
	
	index = threadIdx.x;
	value = d_arr[index];

	d_arr[index] = value * value;

}

int main(int argc, char ** argv)
{

	float * h_array;
	float * d_array;

	size_t totalArraySize;

	totalArraySize = sizeof(float) * THREADS;

	h_array = (float *) malloc(totalArraySize);

	for (size_t i = 0; i < THREADS; i++)
		h_array[i] = i;

	hipMalloc((void **)&d_array, totalArraySize);

	hipMemcpy(d_array, h_array, totalArraySize, hipMemcpyHostToDevice);

	square << <1, THREADS >> > (d_output, d_input);

	hipMemcpy(h_array, d_array, totalArraySize, hipMemcpyDeviceToHost);

	hipFree(d_array);

	for (size_t i = 0; i < THREADS; i++) {
		printf("%8.3f", h_array[i]);
		printf(((i%7)!=6)?"\t":"\n");
	}
	printf("\n");

	free(h_array);

	return 0;

}


