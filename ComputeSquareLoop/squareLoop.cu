
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define SIZE 128
#define THREADS 32


__global__ void squareWithForLoop(float * d_arr, size_t maxLoop, size_t increment)
{

	float value;
	size_t index;
	size_t i;

	index = threadIdx.x;

	for (i = 0; i < maxLoop; i++) {
		value = d_arr[index];
		d_arr[index] = value * value;
		index += increment;
	}

}

int main(int argc, char ** argv) 
{

	float * h_array;
	float * d_array;

	size_t totalArraySize;

	totalArraySize = sizeof(float) * SIZE;

	h_array = (float *)malloc(totalArraySize);

	for (size_t i = 0; i < SIZE; i++)
		h_array[i] = i;

	hipMalloc((void **)&d_array, totalArraySize);

	hipMemcpy(d_array, h_array, totalArraySize, hipMemcpyHostToDevice);

	squareWithForLoop<<<1, THREADS >>> (d_array, SIZE/THREADS, THREADS);

	hipMemcpy(h_array, d_array, totalArraySize, hipMemcpyDeviceToHost);

	hipFree(d_array);

	for (size_t i = 0; i < SIZE; i++) {
		printf("%8.3f", h_array[i]);
		printf(((i % 7) != 6) ? "\t" : "\n");
	}
	printf("\n");

	free(h_array);

	return 0;

}

