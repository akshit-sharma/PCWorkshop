
#include "hip/hip_runtime.h"


#include <stdio.h>

#define NUM_BLOCKS 16
#define THREADS 1

__global__ void helloWorld()
{
	printf("Hello Worlds! I'm a thread in block %d \n", blockIdx.x);
}

int main(int argc, char ** argv)
{

	helloWorld <<< NUM_BLOCKS, THREADS >>> ();
	
	hipDeviceSynchronize();

	printf("That's all\n");

	return 0;

}


